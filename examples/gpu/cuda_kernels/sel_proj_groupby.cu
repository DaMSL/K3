
#include <hip/hip_runtime.h>
struct I {
  int s_id;
  int s_age;
  float s_wage;
};

struct O {
  float s_wage;
};

extern "C" __global__ void
queryplan(I *g_idata, O *g_odata, unsigned int n)
{
    __shared__ float sdata[256];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(256*2) + threadIdx.x;

    float mySum = (i < n) ? g_idata[i].s_wage : 0;

    if (i + 256 < n)
        mySum += g_idata[i+256].s_wage;

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem

    __syncthreads();

    if ((tid < 128))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2)
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if (( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) 
      g_odata[blockIdx.x].s_wage = mySum;
    else {
      int i = blockDim.x * blockIdx.x + tid;
      if (i < n)
      	g_odata[i].s_wage = 0;
    }
}
