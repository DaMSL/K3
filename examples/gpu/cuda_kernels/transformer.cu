
#include <hip/hip_runtime.h>
struct R_elem 
{ 
  int elem;
}; 
__global__ void 
transformer(const R_elem *A, R_elem *C, size_t numElements) 
{                                                      
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;    
  if (i < numElements)                                 
    C[i].elem = A[i].elem + i;                         
}                                                      
