#include "hip/hip_runtime.h"
extern "C" {
struct I {
  int s_id;
  int s_age;
  float s_wage;
};

struct O {
  int s_id;
  float s_wage;
};

__global__ 
void select(const I *A, O *C, size_t n)      
{
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n){                                     
    C[i].s_id = A[i].s_id;                       
    C[i].s_wage = A[i].s_wage ;                   
  }                                           
} 
}
