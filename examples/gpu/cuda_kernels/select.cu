#include "hip/hip_runtime.h"
struct R {                                          
  int elem;                                            
};                                                   
extern "C" __global__                                          
void select(const R *input, R *out_idx, size_t n)      
{
  __shared__ int temp[512];
  size_t thid = threadIdx.x;
  size_t start = 2 * blockIdx.x * blockDim.x;
  temp[2 * thid] = ((start + 2 * thid < n) && (input[start + 2 * thid].elem > 6 ) ? 1: 0);
  temp[2 * thid + 1] = ((start + 2 * thid + 1 < n) && (input[start + 2 * thid + 1].elem > 6 ) ? 1: 0);
  for (int stride = 1; stride <= blockDim.x; stride <<= 1){
    __syncthreads();
    int index = (thid + 1) * stride * 2 - 1;
    if (index < 2 * blockDim.x){
      temp[index] += temp[index - stride];
    }
  }
  if (thid == 0){
    temp[2 * blockDim.x - 1] = 0;
  }
  for (int stride = blockDim.x ; stride >= 1; stride >>= 1){
    __syncthreads();
    int index = (thid + 1) * stride * 2 - 1;
    if (index  < 2 * blockDim.x){
      int t = temp[index - stride];
      temp[index - stride] = temp[index];
      temp[index] += t;
    }
  }  
  __syncthreads();
  temp[2 * thid] = temp[2 * thid + 1] ;
  temp[2 * thid + 1] = (2 * thid + 2 < 2 * blockDim.x ?
     temp[2 * thid + 2] : temp[2 * thid + 1] + (input[2 * thid + 1 + start].elem > 6 ? 1: 0) ) ;
  __syncthreads();
  out_idx[start + 2*thid].elem = -1;
  out_idx[start + 2*thid + 1].elem = -1;
  __syncthreads();
  if (thid != 0 && temp[2 * thid] > temp[2 * thid - 1] ){
    out_idx[start + temp[2 * thid] - 1].elem = 2 * thid;
  } 
  if (thid == 0 && temp[2 * thid] > 0){
   out_idx[start + temp[2 * thid] - 1].elem = 2 * thid;
  } 
  if (temp[2 * thid + 1] > temp[2 * thid]){
    out_idx[start + temp[2 * thid + 1] - 1].elem = 2 * thid + 1;
  } 
} 

