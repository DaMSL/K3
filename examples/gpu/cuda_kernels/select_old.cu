#include "hip/hip_runtime.h"
struct R {                                          
  int elem;                                            
};                                                   
extern "C" __global__                                          
void select(const R *input, R *out_idx, size_t n)      
{
  __shared__ int temp[512];
  size_t thid = threadIdx.x;
  size_t start = 2 * blockIdx.x * blockDim.x;
  // SELECT ELEM > 6
  temp[2 * thid] = ((start + 2 * thid < n) && (input[start + 2 * thid].elem > 6 ) ? 1: 0);
  temp[2 * thid + 1] = ((start + 2 * thid + 1 < n) && (input[start + 2 * thid + 1].elem > 6 ) ? 1: 0);
  
  // perform inclusive scan on temp (3 steps)
  // step 1. reduction 
  for (int stride = 1; stride <= blockDim.x; stride <<= 1){
    __syncthreads();
    int index = (thid + 1) * stride * 2 - 1;
    if (index < 2 * blockDim.x){
      temp[index] += temp[index - stride];
    }
  }
  // step 2. clear last element
  if (thid == 0){
    temp[2 * blockDim.x - 1] = 0;
  }
  // step 3: traverse down the tree
  for (int stride = blockDim.x ; stride >= 1; stride >>= 1){
    __syncthreads();
    int index = (thid + 1) * stride * 2 - 1;
    if (index  < 2 * blockDim.x){
      int t = temp[index - stride];
      temp[index - stride] = temp[index];
      temp[index] += t;
    }
  }  
  __syncthreads();
  // inclusive scan -> exclusive scan
  temp[2 * thid] = temp[2 * thid + 1] ;
  temp[2 * thid + 1] = (2 * thid + 2 < 2 * blockDim.x ?
     temp[2 * thid + 2] : temp[2 * thid + 1] + (input[2 * thid + 1 + start].elem > 6 ? 1: 0) ) ;
  __syncthreads();
  // set all location to -1 (meaning not selected)
  out_idx[start + 2*thid].elem = -1;
  out_idx[start + 2*thid + 1].elem = -1;
  __syncthreads();
  // write index to the right location (overwrite -1)
  if (thid != 0 && temp[2 * thid] > temp[2 * thid - 1] ){
    out_idx[start + temp[2 * thid] - 1].elem = 2 * thid;
  } 
  if (thid == 0 && temp[2 * thid] > 0){
   out_idx[start + temp[2 * thid] - 1].elem = 2 * thid;
  } 
  if (temp[2 * thid + 1] > temp[2 * thid]){
    out_idx[start + temp[2 * thid + 1] - 1].elem = 2 * thid + 1;
  } 
} 

