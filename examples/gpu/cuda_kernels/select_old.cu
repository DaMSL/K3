#include "hip/hip_runtime.h"
struct R {                                          
  int elem;                                            
};                                                   
extern "C" __global__                                          
void select(const R *input, R *out_idx, size_t n)      
{
  __shared__ int temp[512];
  size_t thid = threadIdx.x;
  size_t start = 2 * blockIdx.x * blockDim.x;
  temp[2 * thid] = ((start + 2 * thid < n) && (input[start + 2 * thid].elem < 5) ? 1: 0);
  temp[2 * thid + 1] = ((start + 2 * thid + 1 < n) && (input[start + 2 * thid + 1].elem < 5) ? 1: 0);
  for (int stride = 1; stride <= blockDim.x; stride <<= 1){
    __syncthreads();
    int index = (thid + 1) * stride * 2 - 1;
    if (index < 2 * blockDim.x){
      temp[index] += temp[index - stride];
    }
  }
  if (thid == 0){
    temp[2 * blockDim.x - 1] = 0;
  }
  for (int stride = blockDim.x ; stride >= 1; stride >>= 1){
    __syncthreads();
    int index = (thid + 1) * stride * 2 + 1;
    if (index  < 2 * blockDim.x){
      int t = temp[index - stride];
      temp[index - stride] = temp[index];
      temp[index] += t;
    }
  }
  __syncthreads();
  out_idx[2 * thid + start].elem = temp[2 * thid + 1] ;
  out_idx[2 * thid + start + 1].elem = (2 * thid + 2 < 2 * blockDim.x ? temp[2 * thid + 2] : temp[2 * thid + 1] + input[2 * thid + 1 + start].elem) ;
}



 
