
#include <hip/hip_runtime.h>
struct I {
  int s_id;
  int s_age;
  double s_wage;
};

struct O {
  int s_id;
  double s_wage;
};

extern "C" __global__ 
void select_proj(const I *A, O *C, size_t n)      
{
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) { 
    if (A[i].s_wage > 1400000.0) {                                     
      C[i].s_id = A[i].s_id;                       
      C[i].s_wage = A[i].s_wage;
    }
    else {
      C[i].s_id = 0;
      C[i].s_wage = 0.0;                   
    }
  }                                           
}
